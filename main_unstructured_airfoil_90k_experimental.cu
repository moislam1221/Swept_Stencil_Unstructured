#include "hip/hip_runtime.h"
using namespace std; 
#include "inttypes.h"
#include <vector>
#include <set>
#include <map>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include "initialize.h"
#include "structs.h"
#include "helper.h"
#include "mesh.h"
#include "readFromFile.h"
#include "seeds.h"
#include "subdomains.h"
#include "toDevice.h"
#include "global.h"
#include "swept_update_jacobi.h"
#include "swept_update_jacobi_solutionOnly.h"
#include "residual.h"
#include "matrix.h"
#include "print.h"
#include "debug.h"

//# define PRINT_SOLUTION
# define SAVE_SOLUTION

int main (int argc, char * argv[]) 
{
	// Define number of dofs for the mesh
	uint32_t Ndofs = 91936; // 5876; // this is for the coarse airfoil mesh condensed
	uint32_t numSweptCycles = 10000;
	
	// Define the linear system Ax = b

	// Initialize the linear system and allocate matrix data structures (indexPtr, nodeNeighbors, offdiags) 
	linearSystem matrix;
	matrix.Ndofs = Ndofs;
	initializeAndLoadMatrixFromDirectory(matrix, "Unstructured_Mesh/Airfoil_Mesh_90k/Matrix_Data");   

	for (int i = 0; i < matrix.Ndofs; i++) {
	//	printf("rhs[%d] = %f\n", i, matrix.rhs[i]);
	}

	// Allocate matrix data structures to the GPU (is this even necessary? - yes! used by global memory solution) 
	linearSystemDevice matrix_d;
  	allocateMatrixDevice(matrix_d, matrix);
    copyMatrixDevice(matrix_d, matrix);
	
	/*************** GLOBAL MEMORY START **************************/

	printf("==================== GLOBAL MEMORY ALGORITHM =========================\n");
    
    // Create solution containers on the CPU
    float *du0 = new float[Ndofs];
    float *du1 = new float[Ndofs];
    initializeToZeros(du0, Ndofs);
    initializeToZeros(du1, Ndofs);
    // Create solution containers on the GPU
    float *du0_d;
    float *du1_d;
    hipMalloc(&du0_d, sizeof(float) * Ndofs);
    hipMalloc(&du1_d, sizeof(float) * Ndofs);
    hipMemcpy(du0_d, du0, sizeof(float) * Ndofs, hipMemcpyHostToDevice);
    hipMemcpy(du1_d, du1, sizeof(float) * Ndofs, hipMemcpyHostToDevice);

	// Initial L2 residual
	float residual, residualInit;
	residual = computeL2Residual(du0_d, matrix_d);
	residualInit = residual;
	printf("The initial residual is %f\n", residual);
	
	// Initialize solution and residual norm variables
	float * solutionGM = new float[matrix.Ndofs];
	float residualGM;

	// Number of total Jacobi iterations to perform
	uint32_t numIterations = numSweptCycles * 12;

	// Perform global memory iterations
	float globalTime;	
	hipEvent_t start_g, stop_g;
	hipEventCreate(&start_g);
	hipEventCreate(&stop_g);
	hipEventRecord(start_g, 0); 
    // globalMemorySolve2D(du1_d, du0_d, matrix_d, N, numIterations);
    globalMemorySolveUnstructured(du1_d, du0_d, matrix_d, numIterations);
	hipEventRecord(stop_g, 0);
	hipEventSynchronize(stop_g);
	hipEventElapsedTime(&globalTime, start_g, stop_g); 
	
	// Initial L2 residual
	if (numIterations % 2 == 0) { 
		residual = computeL2Residual(du0_d, matrix_d);
	}
	else if (numIterations % 2 == 1) { 
		residual = computeL2Residual(du1_d, matrix_d);
	}
	residualGM = residual;
	printf("GLOBAL MEMORY ITERATIONS: The initial residual is %f but the final residual after %d iterations is %f\n", residualInit, numIterations, residual);

	// Print solution
	if (numIterations % 2 == 0) { 
		solutionGM = du0_d;
	}
	else if (numIterations % 2 == 1) { 
		solutionGM = du1_d;
	}
	printf("Number of Iterations = %d\n", numIterations);

	// SHARED MEMORY START 
	
	printf("==================== SHARED MEMORY ALGORITHM =========================\n");
	
	// Initialize iteration level
	uint32_t * iterationLevel, * iterationLevel_d, * iterationLevelOutput_d;
	iterationLevel = new uint32_t[Ndofs];
	initializeToZerosInt(iterationLevel, Ndofs);
	// Iteration Level
    hipMalloc(&iterationLevel_d, sizeof(uint32_t) * Ndofs);
	hipMemcpy(iterationLevel_d, iterationLevel, sizeof(uint32_t) * Ndofs, hipMemcpyHostToDevice);
    hipMalloc(&iterationLevelOutput_d, sizeof(uint32_t) * Ndofs);
	hipMemcpy(iterationLevelOutput_d, iterationLevel, sizeof(uint32_t) * Ndofs, hipMemcpyHostToDevice);
	
	float * evenSolutionBuffer_d, * oddSolutionBuffer_d, * solution_d;
	float * evenSolutionBufferOutput_d, * oddSolutionBufferOutput_d;
	hipMalloc(&evenSolutionBuffer_d, sizeof(float) * matrix.Ndofs);
	hipMalloc(&oddSolutionBuffer_d, sizeof(float) * matrix.Ndofs);
	hipMalloc(&evenSolutionBufferOutput_d, sizeof(float) * matrix.Ndofs);
	hipMalloc(&oddSolutionBufferOutput_d, sizeof(float) * matrix.Ndofs);
	hipMalloc(&solution_d, sizeof(float) * matrix.Ndofs);
	uint32_t threadsPerBlock = 256;
	uint32_t numBlocks = ceil((float)matrix.Ndofs / threadsPerBlock);
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(evenSolutionBuffer_d, matrix.Ndofs);	
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(oddSolutionBuffer_d, matrix.Ndofs);	
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(evenSolutionBufferOutput_d, matrix.Ndofs);	
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(oddSolutionBufferOutput_d, matrix.Ndofs);	
	
	// Number of Jacobi iterations to perform in half swept cycle
	uint32_t numJacobiIters = 6;
	
	// DIRECTORY CONTAINING PARTITION AND LEVEL INFORMATION
	std::string PARENT_DIRECTORY = "Unstructured_Mesh/Airfoil_Mesh_90k/";

	// Temporary pointers for swapping	
	float * tmp;
	uint32_t * tmpInt;
	
	/*************** STAGE 1: UPPER PYRAMIDAL STAGE ************************/

	printf("==================== PERFORMING UPPER PYRAMIDAL PARTITIONING =========================\n");
	
	// INITIALIZE
	meshPartitionForStage upperPyramidal;
	upperPyramidal.numSubdomains = 250; // 25

	// SUBDOMAINS
	readSubdomainAndIterationFromFile2(upperPyramidal, PARENT_DIRECTORY, 0);	

	// HOST
	createHaloRegions(upperPyramidal, matrix);
	createTerritoriesHost(upperPyramidal);
	constructLocalMatricesHost(upperPyramidal, matrix);
	
	// DEVICE
	meshPartitionForStageDevice upperPyramidal_d;
	allocatePartitionDevice(upperPyramidal_d, upperPyramidal, Ndofs);
	copyPartitionDevice(upperPyramidal_d, upperPyramidal, Ndofs);
	
	// JACOBI
	// V1 kernel (non-overlapping)
	determineSharedMemoryAllocationSolutionOnly(upperPyramidal);
	// determineSharedMemoryAllocationSolutionOnly(upperPyramidal);
	stageAdvanceJacobiPerformanceSolutionOnly<<<upperPyramidal.numSubdomains, 512, upperPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, upperPyramidal_d, 0, numJacobiIters);
	//determineSharedMemoryAllocationSolutionOnly(upperPyramidal);
	//stageAdvanceJacobiPerformanceSolutionOnly<<<upperPyramidal.numSubdomains, 512, upperPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, upperPyramidal_d, 0, numJacobiIters);
	// V2 kernel (overlapping)
	// stageAdvanceJacobiPerformanceV2Overlap<<<upperPyramidal.numSubdomains, 512, upperPyramidal.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, upperPyramidal_d, 0, numJacobiIters, Ndofs);
	// tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
	// tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
	// tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;

	// POSTPROCESSING	
	assembleSolutionFromBuffers<<<numBlocks, threadsPerBlock>>>(solution_d, evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, Ndofs);
	hipMemcpy(iterationLevel, iterationLevel_d, sizeof(uint32_t) * Ndofs, hipMemcpyDeviceToHost);	
	uint32_t * iterationLevelUpperPyramidal = new uint32_t[Ndofs];
	hipMemcpy(iterationLevelUpperPyramidal, iterationLevel_d, sizeof(uint32_t) * Ndofs, hipMemcpyDeviceToHost);	

#ifdef PRINT_SOLUTION
	printf("================NUMBER OF ITERATIONS PERFORMED IN SHARED==============\n");
	printDeviceArrayInt(iterationLevel_d, Ndofs);
	printf("\n================SIMILARITY TO GLOBAL==============\n");
	printDeviceSimilarity1D(solution_d, solutionGM, 1e-6, Ndofs);
	printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);
#endif
	// printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);

#ifdef SAVE_SOLUTION
	// Save iteration level	
  	ofstream iterationLevelFile;
	iterationLevelFile.open(PARENT_DIRECTORY + "iteration_output_1.txt");
	for (int i = 0; i < matrix.Ndofs; i++) {
  		iterationLevelFile << iterationLevel[i];
  		iterationLevelFile << "\n";
	}
  	iterationLevelFile.close();
#endif

	hipError_t err;
	err = hipDeviceSynchronize();
	printf("%d\n", err);
	if (hipSuccess != err) {
		fprintf(stderr, "failed at %s\n", hipGetErrorString(err));
		exit(-1);
	}
	
    residual = computeL2Residual(solution_d, matrix_d);
	printf("The residual after stage 1 is %f\n", residual);
	
	/*************** STAGE 2: BRIDGE STAGE ************************/
	
	printf("==================== PERFORMING BRIDGE PARTITIONING =========================\n");

	// INITIALIZE
	meshPartitionForStage bridge;
	bridge.numSubdomains = 397; // 397; // 35

	// SUBDOMAINS
	readSubdomainAndIterationFromFile2(bridge, PARENT_DIRECTORY, 1);	
	
	// HOST
	createHaloRegions(bridge, matrix);
	createTerritoriesHost(bridge);
	constructLocalMatricesHost(bridge, matrix);
	
	// DEVICE
	meshPartitionForStageDevice bridge_d;
	allocatePartitionDevice(bridge_d, bridge, Ndofs);
	copyPartitionDevice(bridge_d, bridge, Ndofs);
	
	// JACOBI
	determineSharedMemoryAllocationSolutionOnly(bridge);
	// V1 kernel (non-overlapping)
	// stageAdvanceJacobiPerformance<<<bridge.numSubdomains, 512, bridge.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, bridge_d, 0, numJacobiIters);
	// V2 kernel (overlapping)
	copySolutionToOutput<<<bridge.numSubdomains, 512>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, Ndofs);
	stageAdvanceJacobiPerformanceV2OverlapExperimentalSolutionOnly<<<bridge.numSubdomains, 512, bridge.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, bridge_d, 0, numJacobiIters, Ndofs);
	hipDeviceSynchronize();
	tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
	tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
	tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;
	
	// POSTPROCESSING	
	hipDeviceSynchronize();
	assembleSolutionFromBuffers<<<numBlocks, threadsPerBlock>>>(solution_d, evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, Ndofs);
	hipDeviceSynchronize();
	hipMemcpy(iterationLevel, iterationLevel_d, sizeof(uint32_t) * Ndofs, hipMemcpyDeviceToHost);	
	hipDeviceSynchronize();
	uint32_t * iterationLevelBridge = new uint32_t[Ndofs];
	hipMemcpy(iterationLevelBridge, iterationLevel_d, sizeof(uint32_t) * Ndofs, hipMemcpyDeviceToHost);	

#ifdef PRINT_SOLUTION
	printf("================NUMBER OF ITERATIONS PERFORMED IN SHARED==============\n");
	printDeviceArrayInt(iterationLevel_d, Ndofs);
	printf("\n================SIMILARITY TO GLOBAL==============\n");
	printDeviceSimilarity1D(solution_d, solutionGM, 1e-6, Ndofs);
	printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);
#endif
	printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);

#ifdef SAVE_SOLUTION
	// Save iteration level	
	iterationLevelFile.open(PARENT_DIRECTORY + "iteration_output_2.txt");
	for (int i = 0; i < matrix.Ndofs; i++) {
  		iterationLevelFile << iterationLevel[i];
  		iterationLevelFile << "\n";
	}
  	iterationLevelFile.close();
#endif
		
	err = hipDeviceSynchronize();
	printf("%d\n", err);
	if (hipSuccess != err) {
		fprintf(stderr, "failed at %s\n", hipGetErrorString(err));
		exit(-1);
	}
 	
	residual = computeL2Residual(solution_d, matrix_d);
	printf("The residual after stage 2 is %f\n", residual);
	
	/*************** STAGE 3: LOWER PYRAMIDAL STAGE ************************/
	
	printf("==================== PERFORMING LOWER PYRAMIDAL PARTITIONING =========================\n");

	// INITIALIZE
	meshPartitionForStage lowerPyramidal;
	lowerPyramidal.numSubdomains = 309; // 22;

	// SUBDOMAINS
	readSubdomainAndIterationFromFile2(lowerPyramidal, PARENT_DIRECTORY, 2);	
	
	// HOST
	createHaloRegions(lowerPyramidal, matrix);
	createTerritoriesHost(lowerPyramidal);
	constructLocalMatricesHost(lowerPyramidal, matrix);

	// DEVICE
	meshPartitionForStageDevice lowerPyramidal_d;
	allocatePartitionDevice(lowerPyramidal_d, lowerPyramidal, Ndofs);
	copyPartitionDevice(lowerPyramidal_d, lowerPyramidal, Ndofs);
	
	// JACOBI
	determineSharedMemoryAllocationSolutionOnly(lowerPyramidal);
	// V1 kernel (non-overlapping)
	stageAdvanceJacobiPerformanceSolutionOnly<<<lowerPyramidal.numSubdomains, 512, lowerPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, lowerPyramidal_d, 0, numJacobiIters); 
	// V2 kernel (overlapping)
	// stageAdvanceJacobiPerformanceV2Overlap<<<lowerPyramidal.numSubdomains, 512, lowerPyramidal.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, lowerPyramidal_d, 0, numJacobiIters, Ndofs);
	// tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
	// tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
	// tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;

	// POSTPROCESSING	
	assembleSolutionFromBuffers<<<numBlocks, threadsPerBlock>>>(solution_d, evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, Ndofs);
	hipMemcpy(iterationLevel, iterationLevel_d, sizeof(uint32_t) * Ndofs, hipMemcpyDeviceToHost);	

#ifdef PRINT_SOLUTION
	printf("================NUMBER OF ITERATIONS PERFORMED IN SHARED==============\n");
	printDeviceArrayInt(iterationLevel_d, Ndofs);
	printf("\n================SIMILARITY TO GLOBAL==============\n");
	printDeviceSimilarity1D(solution_d, solutionGM, 1e-6, Ndofs);
	printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);
#endif
	// printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);

#ifdef SAVE_SOLUTION
	// Save iteration level	
	iterationLevelFile.open(PARENT_DIRECTORY + "iteration_output_3.txt");
	for (int i = 0; i < matrix.Ndofs; i++) {
		// printf("Iteration Level[%d] = %d\n", i, iterationLevel[i]);
  		iterationLevelFile << iterationLevel[i];
  		iterationLevelFile << "\n";
	}
  	iterationLevelFile.close();
#endif

	err = hipDeviceSynchronize();
	printf("%d\n", err);
	if (hipSuccess != err) {
		fprintf(stderr, "failed at %s\n", hipGetErrorString(err));
		exit(-1);
	}

	residual = computeL2Residual(solution_d, matrix_d);
	printf("The residual after stage 3 is %f\n", residual);
	
	/*************** STAGE 4: DUAL BRIDGE STAGE ************************/
	
	printf("==================== PERFORMING DUAL BRIDGE PARTITIONING =========================\n");
	
	// INITIALIZE
	meshPartitionForStage dualBridge;
	dualBridge.numSubdomains = 397; // 35;

	// SUBDOMAINS
	readSubdomainAndIterationFromFile2(dualBridge, PARENT_DIRECTORY, 3);

	// HOST
	createHaloRegions(dualBridge, matrix);
	createTerritoriesHost(dualBridge);
	constructLocalMatricesHost(dualBridge, matrix);
	
	// DEVICE
	meshPartitionForStageDevice dualBridge_d;
	allocatePartitionDevice(dualBridge_d, dualBridge, Ndofs);
	copyPartitionDevice(dualBridge_d, dualBridge, Ndofs);
	
	// JACOBI
	// Perform Jacobi Iteration kernel call
	determineSharedMemoryAllocationSolutionOnly(dualBridge);
	// V1 kernel (non-overlapping)
	// stageAdvanceJacobiPerformance<<<dualBridge.numSubdomains, 512, dualBridge.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, dualBridge_d, 0, 2*numJacobiIters); 
	// V2 kernel (overlapping)
	copySolutionToOutput<<<dualBridge.numSubdomains, 512>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, Ndofs);
	stageAdvanceJacobiPerformanceV2OverlapExperimentalSolutionOnly<<<dualBridge.numSubdomains, 512, dualBridge.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, dualBridge_d, 0, numJacobiIters, Ndofs);
	tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
	tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
	tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;
	
	// POSTPROCESSING	
	assembleSolutionFromBuffers<<<numBlocks, threadsPerBlock>>>(solution_d, evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, Ndofs);
	hipMemcpy(iterationLevel, iterationLevel_d, sizeof(uint32_t) * Ndofs, hipMemcpyDeviceToHost);	

#ifdef PRINT_SOLUTION
	printf("================NUMBER OF ITERATIONS PERFORMED IN SHARED==============\n");
	printDeviceArrayInt(iterationLevel_d, Ndofs);
	printf("\n================SIMILARITY TO GLOBAL==============\n");
	printDeviceSimilarity1D(solution_d, solutionGM, 1e-6, Ndofs);
	printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);
#endif
	// printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);

#ifdef SAVE_SOLUTION
	// Save iteration level	
	iterationLevelFile.open(PARENT_DIRECTORY + "iteration_output_4.txt");
	for (int i = 0; i < matrix.Ndofs; i++) {
		// printf("Iteration Level[%d] = %d\n", i, iterationLevel[i]);
  		iterationLevelFile << iterationLevel[i];
  		iterationLevelFile << "\n";
	}
  	iterationLevelFile.close();
#endif

	err = hipDeviceSynchronize();
	printf("%d\n", err);
	if (hipSuccess != err) {
		fprintf(stderr, "failed at %s\n", hipGetErrorString(err));
		exit(-1);
	}

	residual = computeL2Residual(solution_d, matrix_d);
	printf("The residual after stage 4 is %f\n", residual);
	
	/*************** SUBDOMAIN CONSTRUCTION COMPLETE - PERFORM ACTUAL ITERATIONS ON GPU ************************/
	
	// Initialize containers for solution and iteration variables
	threadsPerBlock = 128;
	numBlocks = ceil((float)matrix.Ndofs / threadsPerBlock);
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(evenSolutionBuffer_d, matrix.Ndofs);	
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(oddSolutionBuffer_d, matrix.Ndofs);	
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(evenSolutionBufferOutput_d, matrix.Ndofs);	
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(oddSolutionBufferOutput_d, matrix.Ndofs);	
	initializeToZerosDevice<<<numBlocks, threadsPerBlock>>>(solution_d, matrix.Ndofs);	
    initializeToZerosInt(iterationLevel, matrix.Ndofs);
    hipMemcpy(iterationLevel_d, iterationLevel, sizeof(uint32_t) * Ndofs, hipMemcpyHostToDevice);
    hipMemcpy(iterationLevelOutput_d, iterationLevel, sizeof(uint32_t) * Ndofs, hipMemcpyHostToDevice);
	uint32_t minJacobiIters = 0;
	uint32_t maxJacobiIters = 0;
	uint32_t maxJacobiShift = 0;

	// Initialize CUDA Event timers
	float time_total_1 = 0;
	float time_total_2 = 0;
	float time_total_3 = 0;
	float time_total_4 = 0;
	float time_total_5 = 0;	
	float time_stage_1, time_stage_2, time_stage_3, time_stage_4;	
	hipEvent_t start_1, stop_1, start_2, stop_2, start_3, stop_3, start_4, stop_4, start_5, stop_5;
	hipEventCreate(&start_1);
	hipEventCreate(&stop_1);
	hipEventCreate(&start_2);
	hipEventCreate(&stop_2);
	hipEventCreate(&start_3);
	hipEventCreate(&stop_3);
	hipEventCreate(&start_4);
	hipEventCreate(&stop_4);
	hipEventCreate(&start_5);
	hipEventCreate(&stop_5);
	
	printf("======================= CYCLE START =========================================\n");

	for (int sweptIteration = 0; sweptIteration < numSweptCycles; sweptIteration++) {

		// Print cycle number
		printf("CYCLE %d\n", sweptIteration);

		if (sweptIteration > 0) {
			maxJacobiShift += 12;
		}

		// Set number of Jacobi iterations for first two stages
		maxJacobiIters += 6;

		// STAGE 1: UPPER PYRAMIDAL
		hipEventRecord(start_1, 0);
		// V1 kernel (non-overlapping)
		stageAdvanceJacobiPerformanceSolutionOnly<<<upperPyramidal.numSubdomains, 512, upperPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, upperPyramidal_d, minJacobiIters, maxJacobiIters, maxJacobiShift);
		// stageAdvanceJacobiPerformanceSolutionOnly<<<upperPyramidal.numSubdomains, 512, upperPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, upperPyramidal_d, minJacobiIters, maxJacobiIters, maxJacobiShift);
		// V2 kernel (overlapping)
		// copySolutionToOutput<<<upperPyramidal.numSubdomains, 512>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, Ndofs);
		// stageAdvanceJacobiPerformanceV2OverlapExperimental<<<upperPyramidal.numSubdomains, 512, upperPyramidal.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, upperPyramidal_d, 0, numJacobiIters, Ndofs);
		// tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
		// tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
		// tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;
		//
		hipEventRecord(stop_1, 0);
		hipEventSynchronize(stop_1);
		hipEventElapsedTime(&time_stage_1, start_1, stop_1);
		time_total_1 += time_stage_1;
		//printf("STAGE 1\n");
		//printDeviceArrayInt(iterationLevel_d, Ndofs);

		if (sweptIteration > 0) {
			minJacobiIters += 6;
		}
	
		// STAGE 2: BRIDGE STAGE
		hipEventRecord(start_2, 0);
		// V2 kernel (overlapping)
		copySolutionToOutput<<<numBlocks, 256>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, Ndofs);
		stageAdvanceJacobiPerformanceV2OverlapExperimentalSolutionOnly<<<bridge.numSubdomains, 512, bridge.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, bridge_d, minJacobiIters, maxJacobiIters, Ndofs, maxJacobiShift);
		tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
		tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
		tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;
		//
		hipEventRecord(stop_2, 0);
		hipEventSynchronize(stop_2);
		hipEventElapsedTime(&time_stage_2, start_2, stop_2);
		time_total_2 += time_stage_2;
		//printf("STAGE 2\n");
		//printDeviceArrayInt(iterationLevel_d, Ndofs);

		// Set number of Jacobi iterations for second two stages
		maxJacobiIters += 6;

		// STAGE 3: LOWER PYRAMIDAL
		hipEventRecord(start_3, 0);
		// V1 kernel (non-overlapping)
		// stageAdvanceJacobiPerformance<<<lowerPyramidal.numSubdomains, 512, lowerPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, lowerPyramidal_d, minJacobiIters, maxJacobiIters, maxJacobiShift);
		stageAdvanceJacobiPerformanceSolutionOnly<<<lowerPyramidal.numSubdomains, 512, lowerPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, lowerPyramidal_d, minJacobiIters, maxJacobiIters, maxJacobiShift);
		// V2 kernel (overlapping)
		// stageAdvanceJacobiPerformanceV2Overlap<<<lowerPyramidal.numSubdomains, 512, lowerPyramidal.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, lowerPyramidal_d, minJacobiIters, maxJacobiIters, Ndofs);
		// tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
		// tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
		// tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;

		//
		hipEventRecord(stop_3, 0);
		hipEventSynchronize(stop_3);
		hipEventElapsedTime(&time_stage_3, start_3, stop_3);
		time_total_3 += time_stage_3;
		
		// Set number of Jacobi iterations for second two stages
		minJacobiIters += 6;
		
		// Dual Bridge
		hipEventRecord(start_4, 0);
		// V2 kernel (overlapping)
		copySolutionToOutput<<<numBlocks, 256>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d,  iterationLevel_d, iterationLevelOutput_d, Ndofs);
		stageAdvanceJacobiPerformanceV2OverlapExperimentalSolutionOnly<<<dualBridge.numSubdomains, 512, dualBridge.sharedMemorySize>>>(evenSolutionBufferOutput_d, oddSolutionBufferOutput_d, evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, iterationLevelOutput_d, dualBridge_d, minJacobiIters, maxJacobiIters, Ndofs, maxJacobiShift);
		tmp = evenSolutionBuffer_d; evenSolutionBuffer_d = evenSolutionBufferOutput_d; evenSolutionBufferOutput_d = tmp;
		tmp = oddSolutionBuffer_d; oddSolutionBuffer_d = oddSolutionBufferOutput_d; oddSolutionBufferOutput_d = tmp;
		tmpInt = iterationLevel_d; iterationLevel_d = iterationLevelOutput_d; iterationLevelOutput_d = tmpInt;
		// 
		hipEventRecord(stop_4, 0);
		hipEventSynchronize(stop_4);
		hipEventElapsedTime(&time_stage_4, start_4, stop_4);
		time_total_4 += time_stage_4;
	
	}

	// Set number of Jacobi iterations for final fill-in stage
	maxJacobiShift += 6;
	bool finalStage = true; 

	// FINAL STAGE
	hipEventRecord(start_5, 0);
	// V1 kernel (non-overlapping)
	stageAdvanceJacobiPerformanceSolutionOnly<<<upperPyramidal.numSubdomains, 512, upperPyramidal.sharedMemorySize>>>(evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, upperPyramidal_d, minJacobiIters, maxJacobiIters, maxJacobiShift, finalStage);
	//
	hipEventRecord(stop_5, 0);
	hipEventSynchronize(stop_5);
	hipEventElapsedTime(&time_total_5, start_5, stop_5);

#ifdef SAVE_SOLUTION	
	// Print iteration level
	hipMemcpy(iterationLevel, iterationLevel_d, sizeof(uint32_t) * Ndofs, hipMemcpyDeviceToHost);	
	iterationLevelFile.open(PARENT_DIRECTORY + "iteration_output_final.txt");
	for (int i = 0; i < matrix.Ndofs; i++) {
  		iterationLevelFile << iterationLevel[i];
  		iterationLevelFile << "\n";
	}
  	iterationLevelFile.close();
#endif

	// Print information
	printf("\n==================== FINAL INFORMATION =========================\n");
	assembleSolutionFromBuffers<<<numBlocks, threadsPerBlock>>>(solution_d, evenSolutionBuffer_d, oddSolutionBuffer_d, iterationLevel_d, Ndofs);
#ifdef PRINT_SOLUTION
	printf("================NUMBER OF ITERATIONS PERFORMED IN SHARED==============\n");
	printDeviceArrayInt(iterationLevel_d, Ndofs);
	printf("\n================SIMILARITY TO GLOBAL==============\n");
	printDeviceSimilarity1D(solution_d, solutionGM, 1e-6, Ndofs);
	printGlobalAndSharedMatchDevice(solution_d, solutionGM, iterationLevel_d, numIterations, Ndofs);
#endif
	// Compute L2 residual
	residual = computeL2Residual(solution_d, matrix_d);
	printf("========================RESIDUAL=====================================================================\n");
	printf("Global: The initial residual was %f while the final residual is %f\n", residualInit, residualGM);
	printf("Swept: The initial residual was %f while the final residual is %f\n", residualInit, residual);

	// Print times for global and shared memory to complete
	float sweptTime = time_total_1 + time_total_2 + time_total_3 + time_total_4 + time_total_5;
	printf("========================TIME INFORMATION==============================================================\n");
	printf("Global: Elapsed time in ms %f\n", globalTime);
	printf("Swept: Elapsed time in microseconds %f\n", sweptTime);
	printf("Time for upper pyramidal is %f\n", time_total_1);
	printf("Time for bridge is %f\n", time_total_2);
	printf("Time for lower pyramidal is %f\n", time_total_3);
	printf("Time for dual bridge is %f\n", time_total_4);
	printf("Time for final step is %f\n", time_total_5);
	printf("Speedup is %f\n", globalTime / sweptTime);
	// printDeviceArrayInt(iterationLevel_d, Ndofs);
	
}

